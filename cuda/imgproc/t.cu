//Download CImg.h and put it on the same folder.
//in Linux, compile with:
// g++ main.cpp -o imageViewer -lX11 -pthreads


#include <hip/hip_runtime.h>
#include <iostream>

__global__
void kernel(int x, int y, int s, float *img)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < (x*y*s)) img[i] = (img[i] > 125) ? 255 : 0;
}

float* t (float *f, int x, int y, int s) {
  //for error checking
  hipError_t lerror = hipSuccess;

  //Number of pixels
  int imgSize = x*y*s;
  int imgSizeOnMem = imgSize*sizeof(float);
  
  //Image on device
  float *d_img;
  //Alloc memory on device
  lerror = hipMalloc(&d_img, imgSizeOnMem);

  //transfer from host to device:
  lerror = hipMemcpy(d_img, f, imgSizeOnMem, hipMemcpyHostToDevice);

  kernel<<<(imgSize+255)/256, 256>>>(x, y, s, d_img);

  //Image on host
  float *img = new float[imgSize];
  //Alloc memory on host
  //img = (float*)malloc(imgSizeOnMem);

  //transfer from device to host
  lerror = hipMemcpy(img, d_img, imgSizeOnMem, hipMemcpyDeviceToHost);

  return img;
}